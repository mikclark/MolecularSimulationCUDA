#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include <stdio.h>
#include <math.h>


extern "C" {
    __global__ void EnergyOfExistingMolecule(
        const int NTotal,
        const float* x,
        const float* y,
        const float* z,
        const int* types,
        const float* sigma10x10,
        const float* epsilon10x10,
        const float* lengths,
        const float cutoffFactor,
        const int nthMolecule,
        float* cacheEnergy
        )
    {
        __shared__ float localCache[1024];
        unsigned int threadId = blockDim.x * blockIdx.x + threadIdx.x;
        const unsigned int cacheIndex = threadIdx.x;
        const float iX = x[nthMolecule];
        const float iY = y[nthMolecule];
        const float iZ = z[nthMolecule];
        const int iType = types[nthMolecule];
        const float Lx = lengths[0];
        const float Ly = lengths[1];
        const float Lz = lengths[2];

        float dx = 0.0, dy = 0.0, dz = 0.0, dr2 = 0.0, idr6 = 0.0, ljEnergy = 0.0;
        int jType = -1;
        float tempEnergy = 0;
        const float cutoffFactor2 = cutoffFactor * cutoffFactor;
        while (threadId < NTotal)
        {
            // Do not calculate the energy of the nthMolecule with itself
            if (threadId == nthMolecule) {
                threadId += blockDim.x * gridDim.x;
                continue;
            }

            jType = types[threadId];
            // Skip "empty" molecules, defined by a negative " types" value
            if (jType < 0) {
                threadId += blockDim.x * gridDim.x;
                continue;
            }
            const float isigma2 = sigma10x10[iType + 10 * jType] * sigma10x10[iType + 10 * jType];
            const float iepsilon = epsilon10x10[iType + 10 * jType];

            dx = x[threadId] - iX;
            dy = y[threadId] - iY;
            dz = z[threadId] - iZ;
            dx = dx - Lx * round(dx / Lx);
            dx = dx - Ly * round(dy / Ly);
            dx = dx - Lz * round(dz / Lz);
            dr2 = (dx * dx + dy * dy + dz * dz);
            dr2 /= isigma2;
            if (dr2 <= cutoffFactor2) {
                idr6 = 1.0 / (dr2 * dr2 * dr2);
                ljEnergy = 4.0 * iepsilon * idr6 * (idr6 - 1.0);
                tempEnergy += ljEnergy;
            }
            threadId += blockDim.x * gridDim.x;
        }
        localCache[cacheIndex] = tempEnergy;
        __syncthreads();

        int i = blockDim.x / 2;
        while (i != 0)
        {
            if (cacheIndex < i)
                localCache[cacheIndex] += localCache[cacheIndex + i];
            __syncthreads();
            i /= 2;
        }
        if (cacheIndex == 0)
            cacheEnergy[blockIdx.x] = localCache[0];
    }

    __global__ void VectorSum(const int N, const float* v, float* sum)
    {
        __shared__ float chache[1024];
        unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        const unsigned int chacheindex = threadIdx.x;

        float temp = 0;
        while (tid < N)
        {
            temp += v[tid];
            tid += blockDim.x * gridDim.x;
        }
        chache[chacheindex] = temp;
        __syncthreads();

        int i = blockDim.x / 2;
        while (i != 0)
        {
            if (chacheindex < i)
                chache[chacheindex] += chache[chacheindex + i];
            __syncthreads();
            i /= 2;
        }
        if (chacheindex == 0)
            sum[blockIdx.x] = chache[0];
    }

    __global__ void VectorDotProduct (const int N, const float* V1, const float* V2, float* V3)
    {
        __shared__ float chache[1024];
        float temp;
        unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        const unsigned int chacheindex = threadIdx.x;

        while (tid < N)
        {
            temp += V1[tid] * V2[tid];
            tid += blockDim.x * gridDim.x;
        }
        chache[chacheindex] = temp;
        __syncthreads();

        int i = blockDim.x / 2;
        while (i != 0)
        {
            if (chacheindex < i)
                chache[chacheindex] += chache[chacheindex + i];
            __syncthreads();
            i /= 2;
        }
        if (chacheindex == 0)
            V3[blockIdx.x] = chache[0];
    }
}

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n", devProp.major);
    printf("Minor revision number:         %d\n", devProp.minor);
    printf("Name:                          %s\n", devProp.name);
    printf("Total global memory:           %zu\n", devProp.totalGlobalMem);
    printf("Total shared memory per block: %zu\n", devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n", devProp.regsPerBlock);
    printf("Warp size:                     %d\n", devProp.warpSize);
    printf("Maximum memory pitch:          %zu\n", devProp.memPitch);
    printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
    for (int i = 1; i <= 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i - 1]);
    for (int i = 1; i <= 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i - 1]);
    printf("Clock rate:                    %d\n", devProp.clockRate);
    printf("Total constant memory:         %zu\n", devProp.totalConstMem);
    printf("Texture alignment:             %zu\n", devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

int main()
{
    // Number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    // Iterate through devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }

    printf("\nPress any key to exit...");
    char c;
    scanf("%c", &c);

    return 0;
}